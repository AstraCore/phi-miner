/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "device_info.h"
#include <stdio.h>
//#include<bits/stdc++.h>
#include <vector>
#include <string>

void printDeviceProps() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number BLA BLA: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
      printf("  Major: %d Minor: %d\n",prop.major, prop.minor);

  }

}

int getDeviceCount() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    return nDevices;
}

std::vector <std::string> getPropsOfIDevice(int i) {
    std::vector <std::string> props;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    std::string name = prop.name;
    std::string version = std::to_string(prop.major) + "." + std::to_string(prop.minor);
    props.push_back(name);
    props.push_back(version);
    return props;
}

std::vector<int> getMinableDevices() {
    std::vector<int> devices;
    int nDevices = getDeviceCount();
    for(int i=0; i<nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if(prop.major >= 5) {
            devices.push_back(i);
        }
    }
    return devices;
}